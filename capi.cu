#include "cusz.h"

#include "cli/quality_viewer.hh"
#include "cli/timerecord_viewer.hh"
#include "utils/io.hh"
#include "utils/print_gpu.hh"

extern "C" {
void** compress(float* deviceInputPtr, uint8_t* deviceCompressedPtr, int fileSize, float errorBound, size_t* compressedLen)
{
    auto len = fileSize;

    // cusz_header header;
    cusz_header* headerPtr = (cusz_header*)malloc(sizeof(cusz_header));
    uint8_t*    exposed_compressed;
    // size_t      compressed_len;

    float *d_uncompressed;
    
    d_uncompressed = deviceInputPtr;

    /* code snippet for looking at the device array easily */
    auto peek_devdata = [](float* d_arr, size_t num = 20) {
        thrust::for_each(thrust::device, d_arr, d_arr + num, [=] __device__ __host__(const float i) { printf("%f\t", i); });
        printf("\n");
    };

    /* a casual peek */
    printf("peeking uncompressed data, 20 elements\n");
    peek_devdata(d_uncompressed, 20);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // using default
    // cusz_framework* framework = cusz_default_framework();
    // alternatively
    cusz_framework* framework = new cusz_custom_framework{
        .pipeline     = Auto,
        .predictor    = cusz_custom_predictor{.type = LorenzoI},
        .quantization = cusz_custom_quantization{.radius = 512},
        .codec        = cusz_custom_codec{.type = Huffman}};

    cusz_compressor* comp       = cusz_create(framework, FP32);
    cusz_config*     config     = new cusz_config{.eb = errorBound, .mode = Rel};
    cusz_len         uncomp_len = cusz_len{len, 1, 1, 1};
    cusz_len         decomp_len = uncomp_len;

    cusz::TimeRecord compress_timerecord;

    {
        cusz_compress(
            comp, config, d_uncompressed, uncomp_len, &exposed_compressed, compressedLen, headerPtr,
            (void*)&compress_timerecord, stream);

        /* User can interpret the collected time information in other ways. */
        cusz::TimeRecordViewer::view_compression(&compress_timerecord, len * sizeof(float), *compressedLen);

        /* verify header */
        printf("header.%-*s : %x\n", 12, "(addr)", headerPtr);
        printf("header.%-*s : %lu, %lu, %lu\n", 12, "{x,y,z}", headerPtr->x, headerPtr->y, headerPtr->z);
        printf("header.%-*s : %lu\n", 12, "filesize", ConfigHelper::get_filesize(headerPtr));
    }

    /* If needed, User should perform a memcopy to transfer `exposed_compressed` before `compressor` is destroyed. */
    // hipMalloc(&compressed, compressed_len);
    hipMemcpy(deviceCompressedPtr, exposed_compressed, *compressedLen, hipMemcpyDeviceToDevice);

    void** res = (void**)malloc(sizeof(void*) * 3);
    res[0] = (void*) headerPtr;
    res[1] = (void*) comp;
    res[2] = (void*) stream;

    return res;
}

void decompress(uint8_t* deviceCompressedPtr, float* deviceOutputPtr, int fileSize, float errorBound, size_t* compressedLen, void** ptrs)
{
    auto len = fileSize;

    cusz_header* headerPtr = (cusz_header*)ptrs[0];
    // uint8_t*    exposed_compressed;
    // size_t      compressed_len;

    float *d_decompressed;
    
    d_decompressed = deviceOutputPtr;

    auto peek_devdata = [](float* d_arr, size_t num = 20) {
        thrust::for_each(thrust::device, d_arr, d_arr + num, [=] __device__ __host__(const float i) { printf("%f\t", i); });
        printf("\n");
    };

    hipStream_t stream = (hipStream_t)ptrs[2];

    cusz_compressor* comp       = (cusz_compressor*)ptrs[1];
    cusz_len         uncomp_len = cusz_len{len, 1, 1, 1};
    cusz_len         decomp_len = uncomp_len;

    cusz::TimeRecord decompress_timerecord;

    {
        cusz_decompress(
            comp, headerPtr, deviceCompressedPtr, *compressedLen, d_decompressed, decomp_len,
            (void*)&decompress_timerecord, stream);

        cusz::TimeRecordViewer::view_decompression(&decompress_timerecord, len * sizeof(float));
    }

    /* a casual peek */
    printf("peeking decompressed data, 20 elements\n");
    peek_devdata(d_decompressed, 20);

    free(headerPtr);
    cusz_release(comp);
    hipStreamDestroy(stream);
}
}